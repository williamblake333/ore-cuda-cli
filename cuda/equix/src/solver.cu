#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "solver.h"
#include "context.h"
#include "solver_heap.h"
#include <../../hashx/src/hashx_endian.h>
#include <string.h>
#include <stdbool.h>
#include <assert.h>
#include <stdio.h>
#include <stdint.h>

#ifdef _MSC_VER
#pragma warning (disable : 4146) /* unary minus applied to unsigned type */
#endif

#define CLEAR(x) memset(&x, 0, sizeof(x))
#define MAKE_ITEM(bucket, left, right) ((left) << 17 | (right) << 8 | (bucket))
#define ITEM_BUCKET(item) (item) % NUM_COARSE_BUCKETS
#define ITEM_LEFT_IDX(item) (item) >> 17
#define ITEM_RIGHT_IDX(item) ((item) >> 8) & 511
#define INVERT_BUCKET(idx) -(idx) % NUM_COARSE_BUCKETS
#define INVERT_SCRATCH(idx) -(idx) % NUM_FINE_BUCKETS
#define STAGE1_IDX(buck, pos) heap->stage1_indices.buckets[buck].items[pos]
#define STAGE2_IDX(buck, pos) heap->stage2_indices.buckets[buck].items[pos]
#define STAGE3_IDX(buck, pos) heap->stage3_indices.buckets[buck].items[pos]
#define STAGE1_DATA(buck, pos) heap->stage1_data.buckets[buck].items[pos]
#define STAGE2_DATA(buck, pos) heap->stage2_data.buckets[buck].items[pos]
#define STAGE3_DATA(buck, pos) heap->stage3_data.buckets[buck].items[pos]
#define STAGE1_SIZE(buck) heap->stage1_indices.counts[buck]
#define STAGE2_SIZE(buck) heap->stage2_indices.counts[buck]
#define STAGE3_SIZE(buck) heap->stage3_indices.counts[buck]
#define SCRATCH(buck, pos) heap->scratch_ht.buckets[buck].items[pos]
#define SCRATCH_SIZE(buck) heap->scratch_ht.counts[buck]
#define SWAP_IDX(a, b)      \
    do {                    \
        equix_idx temp = a; \
        a = b;              \
        b = temp;           \
    } while(0)
#define CARRY (bucket_idx != 0)
#define BUCK_START 0
#define BUCK_END (NUM_COARSE_BUCKETS / 2 + 1)

typedef uint32_t u32;
typedef stage1_idx_item s1_idx;
typedef stage2_idx_item s2_idx;
typedef stage3_idx_item s3_idx;

__device__ FORCE_INLINE uint64_t hash_value(hashx_ctx* hash_func, equix_idx index) {
	char hash[HASHX_SIZE];
	hashx_exec(hash_func, index, hash);
	return load64(hash);
}

__device__ unsigned int atomicAdd_u16(uint16_t *address, uint16_t val) {
	unsigned int* base_address = (unsigned int*)((char*)address - ((size_t)address & 2));
  unsigned int old, assumed;
  old = *base_address;
  do {
      assumed = old;
      old = atomicCAS(base_address, assumed,
                      (assumed & 0xFFFF0000) | (((assumed & 0xFFFF) + val) & 0xFFFF));
  } while (assumed != old);
  return old;
}

__device__ unsigned int atomicSub_u16(uint16_t *address, uint16_t val) {
	unsigned int* base_address = (unsigned int*)((char*)address - ((size_t)address & 2));
  unsigned int old, assumed;
  old = *base_address;
  do {
      assumed = old;
      old = atomicCAS(base_address, assumed,
                      (assumed & 0xFFFF0000) | (((assumed & 0xFFFF) - val) & 0xFFFF));
  } while (assumed != old);
  return old;
}

__device__ void build_solution_stage1(equix_idx* output, solver_heap* heap, s2_idx root) {
	u32 bucket = ITEM_BUCKET(root);
	u32 bucket_inv = INVERT_BUCKET(bucket);
	u32 left_parent_idx = ITEM_LEFT_IDX(root);
	u32 right_parent_idx = ITEM_RIGHT_IDX(root);
	s1_idx left_parent = STAGE1_IDX(bucket, left_parent_idx);
	s1_idx right_parent = STAGE1_IDX(bucket_inv, right_parent_idx);
	output[0] = left_parent;
	output[1] = right_parent;
	if (!tree_cmp1(&output[0], &output[1])) {
		SWAP_IDX(output[0], output[1]);
	}
}

__device__ void build_solution_stage2(equix_idx* output, solver_heap* heap, s3_idx root) {
	u32 bucket = ITEM_BUCKET(root);
	u32 bucket_inv = INVERT_BUCKET(bucket);
	u32 left_parent_idx = ITEM_LEFT_IDX(root);
	u32 right_parent_idx = ITEM_RIGHT_IDX(root);
	s2_idx left_parent = STAGE2_IDX(bucket, left_parent_idx);
	s2_idx right_parent = STAGE2_IDX(bucket_inv, right_parent_idx);
	build_solution_stage1(&output[0], heap, left_parent);
	build_solution_stage1(&output[2], heap, right_parent);
	if (!tree_cmp2(&output[0], &output[2])) {
		SWAP_IDX(output[0], output[2]);
		SWAP_IDX(output[1], output[3]);
	}
}

__device__ void build_solution(equix_solution* solution, solver_heap* heap, s3_idx left, s3_idx right) {
	build_solution_stage2(&solution->idx[0], heap, left);
	build_solution_stage2(&solution->idx[4], heap, right);
	if (!tree_cmp4(&solution->idx[0], &solution->idx[4])) {
		SWAP_IDX(solution->idx[0], solution->idx[4]);
		SWAP_IDX(solution->idx[1], solution->idx[5]);
		SWAP_IDX(solution->idx[2], solution->idx[6]);
		SWAP_IDX(solution->idx[3], solution->idx[7]);
	}
}

__device__ void solve_stage0(uint64_t* hashes, solver_heap* heap) {
	CLEAR(heap->stage1_indices.counts);
	for (u32 i = 0; i < INDEX_SPACE; ++i) {
		uint64_t value = hashes[i];
		u32 bucket_idx = value % NUM_COARSE_BUCKETS;
		u32 item_idx = STAGE1_SIZE(bucket_idx);
		if (item_idx >= COARSE_BUCKET_ITEMS)
			continue;
		STAGE1_SIZE(bucket_idx) = item_idx + 1;
		STAGE1_IDX(bucket_idx, item_idx) = i;
		STAGE1_DATA(bucket_idx, item_idx) = value / NUM_COARSE_BUCKETS; /* 52 bits */
	}
}

__device__ void hash_stage0i(hashx_ctx* hash_func, uint64_t* out, uint32_t i) {
	uint64_t hash = hash_value(hash_func, i);
	memcpy((char*)out + (i * sizeof(uint64_t)), &hash, sizeof(uint64_t));
}

#define MAKE_PAIRS1                                                           \
    stage1_data_item value = STAGE1_DATA(bucket_idx, item_idx) + CARRY;       \
    u32 fine_buck_idx = value % NUM_FINE_BUCKETS;                             \
    u32 fine_cpl_bucket = INVERT_SCRATCH(fine_buck_idx);                      \
    u32 fine_cpl_size = SCRATCH_SIZE(fine_cpl_bucket);                        \
    for (u32 fine_idx = 0; fine_idx < fine_cpl_size; ++fine_idx) {            \
        u32 cpl_index = SCRATCH(fine_cpl_bucket, fine_idx);                   \
        stage1_data_item cpl_value = STAGE1_DATA(cpl_bucket, cpl_index);      \
        stage1_data_item sum = value + cpl_value;                             \
        assert((sum % NUM_FINE_BUCKETS) == 0);                                \
        sum /= NUM_FINE_BUCKETS; /* 45 bits */                                \
        u32 s2_buck_id = sum % NUM_COARSE_BUCKETS;                            \
        u32 s2_item_id = STAGE2_SIZE(s2_buck_id);                             \
        if (s2_item_id >= COARSE_BUCKET_ITEMS)                                \
            continue;                                                         \
        STAGE2_SIZE(s2_buck_id) = s2_item_id + 1;                             \
        STAGE2_IDX(s2_buck_id, s2_item_id) =                                  \
            MAKE_ITEM(bucket_idx, item_idx, cpl_index);                       \
        STAGE2_DATA(s2_buck_id, s2_item_id) =                                 \
            sum / NUM_COARSE_BUCKETS; /* 37 bits */                           \
    }                                                                         \

__device__ void solve_stage1(solver_heap* heap) {
	CLEAR(heap->stage2_indices.counts);
	for (u32 bucket_idx = BUCK_START; bucket_idx < BUCK_END; ++bucket_idx) {
		u32 cpl_bucket = INVERT_BUCKET(bucket_idx);
		CLEAR(heap->scratch_ht.counts);
		u32 cpl_buck_size = STAGE1_SIZE(cpl_bucket);
		for (u32 item_idx = 0; item_idx < cpl_buck_size; ++item_idx) {
			stage1_data_item value = STAGE1_DATA(cpl_bucket, item_idx);
			u32 fine_buck_idx = value % NUM_FINE_BUCKETS;
			u32 fine_item_idx = SCRATCH_SIZE(fine_buck_idx);
			if (fine_item_idx >= FINE_BUCKET_ITEMS)
				continue;
			SCRATCH_SIZE(fine_buck_idx) = fine_item_idx + 1;
			SCRATCH(fine_buck_idx, fine_item_idx) = item_idx;
			if (cpl_bucket == bucket_idx) {
				MAKE_PAIRS1
			}
		}
		if (cpl_bucket != bucket_idx) {
			u32 buck_size = STAGE1_SIZE(bucket_idx);
			for (u32 item_idx = 0; item_idx < buck_size; ++item_idx) {
				MAKE_PAIRS1
			}
		}
	}
}

#define MAKE_PAIRS2                                                           \
    stage2_data_item value = STAGE2_DATA(bucket_idx, item_idx) + CARRY;       \
    u32 fine_buck_idx = value % NUM_FINE_BUCKETS;                             \
    u32 fine_cpl_bucket = INVERT_SCRATCH(fine_buck_idx);                      \
    u32 fine_cpl_size = SCRATCH_SIZE(fine_cpl_bucket);                        \
    for (u32 fine_idx = 0; fine_idx < fine_cpl_size; ++fine_idx) {            \
        u32 cpl_index = SCRATCH(fine_cpl_bucket, fine_idx);                   \
        stage2_data_item cpl_value = STAGE2_DATA(cpl_bucket, cpl_index);      \
        stage2_data_item sum = value + cpl_value;                             \
        assert((sum % NUM_FINE_BUCKETS) == 0);                                \
        sum /= NUM_FINE_BUCKETS; /* 30 bits */                                \
        u32 s3_buck_id = sum % NUM_COARSE_BUCKETS;                            \
        u32 s3_item_id = STAGE3_SIZE(s3_buck_id);                             \
        if (s3_item_id >= COARSE_BUCKET_ITEMS)                                \
            continue;                                                         \
        STAGE3_SIZE(s3_buck_id) = s3_item_id + 1;                             \
        STAGE3_IDX(s3_buck_id, s3_item_id) =                                  \
            MAKE_ITEM(bucket_idx, item_idx, cpl_index);                       \
        STAGE3_DATA(s3_buck_id, s3_item_id) =                                 \
            sum / NUM_COARSE_BUCKETS; /* 22 bits */                           \
    }                                                                         \

__device__ void solve_stage2(solver_heap* heap) {
	CLEAR(heap->stage3_indices.counts);
	for (u32 bucket_idx = BUCK_START; bucket_idx < BUCK_END; ++bucket_idx) {
		u32 cpl_bucket = INVERT_BUCKET(bucket_idx);
		CLEAR(heap->scratch_ht.counts);
		u32 cpl_buck_size = STAGE2_SIZE(cpl_bucket);
		for (u32 item_idx = 0; item_idx < cpl_buck_size; ++item_idx) {
			stage2_data_item value = STAGE2_DATA(cpl_bucket, item_idx);
			u32 fine_buck_idx = value % NUM_FINE_BUCKETS;
			u32 fine_item_idx = SCRATCH_SIZE(fine_buck_idx);
			if (fine_item_idx >= FINE_BUCKET_ITEMS)
				continue;
			SCRATCH_SIZE(fine_buck_idx) = fine_item_idx + 1;
			SCRATCH(fine_buck_idx, fine_item_idx) = item_idx;
			if (cpl_bucket == bucket_idx) {
				MAKE_PAIRS2
			}
		}
		if (cpl_bucket != bucket_idx) {
			u32 buck_size = STAGE2_SIZE(bucket_idx);
			for (u32 item_idx = 0; item_idx < buck_size; ++item_idx) {
				MAKE_PAIRS2
			}
		}
	}
}

#define MAKE_PAIRS3                                                           \
    stage3_data_item value = STAGE3_DATA(bucket_idx, item_idx) + CARRY;       \
    u32 fine_buck_idx = value % NUM_FINE_BUCKETS;                             \
    u32 fine_cpl_bucket = INVERT_SCRATCH(fine_buck_idx);                      \
    u32 fine_cpl_size = SCRATCH_SIZE(fine_cpl_bucket);                        \
    for (u32 fine_idx = 0; fine_idx < fine_cpl_size; ++fine_idx) {            \
        u32 cpl_index = SCRATCH(fine_cpl_bucket, fine_idx);                   \
        stage3_data_item cpl_value = STAGE3_DATA(cpl_bucket, cpl_index);      \
        stage3_data_item sum = value + cpl_value;                             \
        assert((sum % NUM_FINE_BUCKETS) == 0);                                \
        sum /= NUM_FINE_BUCKETS; /* 15 bits */                                \
        if ((sum & EQUIX_STAGE1_MASK) == 0) {                                 \
            /* we have a solution */                                          \
            s3_idx item_left = STAGE3_IDX(bucket_idx, item_idx);              \
            s3_idx item_right = STAGE3_IDX(cpl_bucket, cpl_index);            \
            build_solution(&output[sols_found], heap, item_left, item_right); \
            if (++(sols_found) >= EQUIX_MAX_SOLS) {                           \
                return sols_found;                                            \
            }                                                                 \
        }                                                                     \
    }                                                                         \

__device__ uint32_t solve_stage3(solver_heap* heap, equix_solution output[EQUIX_MAX_SOLS]) {
	uint32_t sols_found = 0;

	for (u32 bucket_idx = BUCK_START; bucket_idx < BUCK_END; ++bucket_idx) {
		u32 cpl_bucket = -bucket_idx & (NUM_COARSE_BUCKETS - 1);
		CLEAR(heap->scratch_ht.counts);
		u32 cpl_buck_size = STAGE3_SIZE(cpl_bucket);
		for (u32 item_idx = 0; item_idx < cpl_buck_size; ++item_idx) {
			stage3_data_item value = STAGE3_DATA(cpl_bucket, item_idx);
			u32 fine_buck_idx = value % NUM_FINE_BUCKETS;
			u32 fine_item_idx = SCRATCH_SIZE(fine_buck_idx);
			if (fine_item_idx >= FINE_BUCKET_ITEMS)
				continue;
			SCRATCH_SIZE(fine_buck_idx) = fine_item_idx + 1;
			SCRATCH(fine_buck_idx, fine_item_idx) = item_idx;
			if (cpl_bucket == bucket_idx) {
				MAKE_PAIRS3
			}
		}
		if (cpl_bucket != bucket_idx) {
			u32 buck_size = STAGE3_SIZE(bucket_idx);
			for (u32 item_idx = 0; item_idx < buck_size; ++item_idx) {
				MAKE_PAIRS3
			}
		}
	}

	return sols_found;
}

// GPU kernel
__global__ void solve_all_stages_kernel(uint64_t* hashes, solver_heap* heaps, equix_solution* solutions, uint32_t* num_sols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    uint64_t* thread_hashes = hashes + (idx * INDEX_SPACE);
    solver_heap* thread_heap = &heaps[idx];
    equix_solution* thread_solutions = &solutions[idx * EQUIX_MAX_SOLS];
    
    solve_stage0(thread_hashes, thread_heap);
	__syncthreads();
    solve_stage1(thread_heap);
	__syncthreads();
    solve_stage2(thread_heap);
	__syncthreads();
    num_sols[idx] = solve_stage3(thread_heap, thread_solutions);
}
