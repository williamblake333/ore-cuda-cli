#include "hip/hip_runtime.h"
#include <stdint.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>

#include "blake2.h"
#include "hashx_endian.h"

// Constants for Blake2b
static const uint64_t blake2b_IV[8] = {
    UINT64_C(0x6a09e667f3bcc908), UINT64_C(0xbb67ae8584caa73b),
    UINT64_C(0x3c6ef372fe94f82b), UINT64_C(0xa54ff53a5f1d36f1),
    UINT64_C(0x510e527fade682d1), UINT64_C(0x9b05688c2b3e6c1f),
    UINT64_C(0x1f83d9abfb41bd6b), UINT64_C(0x5be0cd19137e2179)
};

// Predefined Sigma Constants
static const uint8_t blake2b_sigma[12][16] = {
    { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
    { 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
    { 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
    { 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
    { 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
    { 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
    { 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
    { 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
    { 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
    { 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
    { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
    { 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 }
};

// Rotate right function
static FORCE_INLINE uint64_t rotr64(const uint64_t w, const unsigned c) {
    return (w >> c) | (w << (64 - c));
}

// Set the last block indicator flag
static FORCE_INLINE void blake2b_set_lastblock(blake2b_state* S) {
    S->f[0] = (uint64_t)-1;
}

// Increment the counter
static FORCE_INLINE void blake2b_increment_counter(blake2b_state* S, uint64_t inc) {
    S->t[0] += inc;
    if (S->t[0] < inc) {
        S->t[1]++;
    }
}

// Initialize the Blake2b state with IVs
static FORCE_INLINE void blake2b_init0(blake2b_state* S) {
    memset(S, 0, sizeof(*S));
    memcpy(S->h, blake2b_IV, sizeof(S->h));
}

// Initialize the Blake2b state with parameters
int hashx_blake2b_init_param(blake2b_state* S, const blake2b_param* P) {
    if (!P || !S) {
        return -1;
    }

    blake2b_init0(S);
    const unsigned char* p = (const unsigned char*)P;
    for (unsigned int i = 0; i < 8; ++i) {
        S->h[i] ^= load64(&p[i * sizeof(S->h[i])]);
    }
    S->outlen = P->digest_length;
    return 0;
}

// G function for Blake2b's F function
#define G(r, i, j, a, b, c, d)                                               \
    do {                                                                     \
        a += b + m[blake2b_sigma[r][i]];                                     \
        d = rotr64(d ^ a, 32);                                               \
        c += d;                                                              \
        b = rotr64(b ^ c, 24);                                               \
        a += b + m[blake2b_sigma[r][j]];                                     \
        d = rotr64(d ^ a, 16);                                               \
        c += d;                                                              \
        b = rotr64(b ^ c, 63);                                               \
    } while (0)

// Round function with the inner rounds of Blake2b
#define ROUND(r)                                                             \
    do {                                                                     \
        G(r,  0,  1, v[0], v[4], v[8], v[12]);                               \
        G(r,  2,  3, v[1], v[5], v[9], v[13]);                               \
        G(r,  4,  5, v[2], v[6], v[10], v[14]);                              \
        G(r,  6,  7, v[3], v[7], v[11], v[15]);                              \
        G(r,  8,  9, v[0], v[5], v[10], v[15]);                              \
        G(r, 10, 11, v[1], v[6], v[11], v[12]);                              \
        G(r, 12, 13, v[2], v[7], v[8], v[13]);                               \
        G(r, 14, 15, v[3], v[4], v[9], v[14]);                               \
    } while (0)

// Compress function for Blake2b
static void blake2b_compress(blake2b_state* S, const uint8_t* block) {
    uint64_t m[16];
    uint64_t v[16];

    for (unsigned int i = 0; i < 16; ++i) {
        m[i] = load64(block + i * sizeof(m[i]));
    }

    for (unsigned int i = 0; i < 8; ++i) {
        v[i] = S->h[i];
    }

    v[8] = blake2b_IV[0];
    v[9] = blake2b_IV[1];
    v[10] = blake2b_IV[2];
    v[11] = blake2b_IV[3];
    v[12] = blake2b_IV[4] ^ S->t[0];
    v[13] = blake2b_IV[5] ^ S->t[1];
    v[14] = blake2b_IV[6] ^ S->f[0];
    v[15] = blake2b_IV[7] ^ S->f[1];

    // 12 rounds
    for (unsigned int i = 0; i < 12; ++i) {
        ROUND(i);
    }

    for (unsigned int i = 0; i < 8; ++i) {
        S->h[i] ^= v[i] ^ v[i + 8];
    }
}

// 4-round version of the Blake2b compress function
static void blake2b_compress_4r(blake2b_state* S, const uint8_t* block) {
    uint64_t m[16];
    uint64_t v[16];

    for (unsigned int i = 0; i < 16; ++i) {
        m[i] = load64(block + i * sizeof(m[i]));
    }

    for (unsigned int i = 0; i < 8; ++i) {
        v[i] = S->h[i];
    }

    v[8] = blake2b_IV[0];
    v[9] = blake2b_IV[1];
    v[10] = blake2b_IV[2];
    v[11] = blake2b_IV[3];
    v[12] = blake2b_IV[4] ^ S->t[0];
    v[13] = blake2b_IV[5] ^ S->t[1];
    v[14] = blake2b_IV[6] ^ S->f[0];
    v[15] = blake2b_IV[7] ^ S->f[1];

    // 4 rounds
    for (unsigned int i = 0; i < 4; ++i) {
        ROUND(i);
    }

    for (unsigned int i = 0; i < 8; ++i) {
        S->h[i] ^= v[i] ^ v[i + 8];
    }
}

// Update function for Blake2b
int hashx_blake2b_update(blake2b_state* S, const void* in, size_t inlen) {
    if (!S || !in || inlen == 0) {
        return 0;
    }

    const uint8_t* pin = (const uint8_t*)in;

    while (inlen > 0) {
        size_t fill = BLAKE2B_BLOCKBYTES - S->buflen;

        if (inlen > fill) {
            memcpy(S->buf + S->buflen, pin, fill);
            blake2b_increment_counter(S, BLAKE2B_BLOCKBYTES);
            blake2b_compress(S, S->buf);
            S->buflen = 0;
            inlen -= fill;
            pin += fill;
        } else {
            memcpy(S->buf + S->buflen, pin, inlen);
            S->buflen += (unsigned int)inlen;
            break;
        }
    }

    return 0;
}

// Finalization function for Blake2b
int hashx_blake2b_final(blake2b_state* S, void* out, size_t outlen) {
    if (!S || !out || outlen < S->outlen || S->f[0] != 0) {
        return -1;
    }

    blake2b_increment_counter(S, S->buflen);
    blake2b_set_lastblock(S);
    memset(S->buf + S->buflen, 0, BLAKE2B_BLOCKBYTES - S->buflen); /* Padding */
    blake2b_compress(S, S->buf);

    for (unsigned int i = 0; i < 8; ++i) {
        store64((uint8_t*)out + sizeof(S->h[i]) * i, S->h[i]);
    }

    return 0;
}

// 4-round version of Blake2b
void hashx_blake2b_4r(const blake2b_param* params, const void* in, size_t inlen, void* out) {
    blake2b_state state;

    if (!params || !in || !out) return;

    blake2b_init0(&state);

    for (unsigned int i = 0; i < 8; ++i) {
        state.h[i] ^= load64(((const uint8_t*)params) + i * sizeof(state.h[i]));
    }

    const uint8_t* pin = (const uint8_t*)in;

    while (inlen > BLAKE2B_BLOCKBYTES) {
        blake2b_increment_counter(&state, BLAKE2B_BLOCKBYTES);
        blake2b_compress_4r(&state, pin);
        inlen -= BLAKE2B_BLOCKBYTES;
        pin += BLAKE2B_BLOCKBYTES;
    }

    memcpy(state.buf, pin, inlen);
    blake2b_increment_counter(&state, inlen);
    blake2b_set_lastblock(&state);
    blake2b_compress_4r(&state, state.buf);

    /* Output hash */
    memcpy(out, state.h, sizeof(state.h));
}
