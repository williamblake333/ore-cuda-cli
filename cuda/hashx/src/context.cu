#include <stdlib.h>
#include <string.h>
#include <../include/hashx.h>
#include "context.h"
#include "compiler.h"
#include "program.h"

#define STRINGIZE_INNER(x) #x
#define STRINGIZE(x) STRINGIZE_INNER(x)

#ifndef HASHX_SALT
#define HASHX_SALT HashX v1
#endif

// The definition remains here
__device__ const blake2b_param hashx_blake2_params = {
    64, 0, 1, 1, 0, 0, 0, 0, { 0 }, STRINGIZE(HASHX_SALT), { 0 }
};

hashx_ctx* hashx_alloc(hashx_type type) {
    hashx_ctx* ctx = NULL;

    // Allocate unified memory for context
    hipError_t err = hipMallocManaged(&ctx, sizeof(hashx_ctx));
    if (err != hipSuccess) {
        return NULL;
    }

    // Initialize pointers to NULL
    ctx->code = NULL;
    ctx->program = NULL;

    // Allocate memory based on the type of context
    if (type & HASHX_COMPILED) {
        if (!hashx_compiler_init(ctx)) {
            hipFree(ctx);
            return NULL;
        }
        ctx->type = HASHX_COMPILED;
    } else {
        err = hipMallocManaged(&ctx->program, sizeof(hashx_program));
        if (err != hipSuccess) {
            hipFree(ctx);
            return NULL;
        }
        ctx->type = HASHX_INTERPRETED;
    }

#ifdef HASHX_BLOCK_MODE
    // Directly initialize the blake2b_param structure
    ctx->params = hashx_blake2_params;
#endif

    return ctx;
}

void hashx_free(hashx_ctx* ctx) {
    if (ctx != NULL && ctx != HASHX_NOTSUPP) {
        if (ctx->type & HASHX_COMPILED) {
            hashx_compiler_destroy(ctx);
        } else {
            hipFree(ctx->program);
        }
        hipFree(ctx);
    }
}
