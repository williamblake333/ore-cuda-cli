#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <vector>
#include "drillx.h"
#include "equix/include/equix.h"
#include "hashx/include/hashx.h"
#include "equix/src/context.h"
#include "equix/src/solver.h"
#include "equix/src/solver_heap.h"
#include "hashx/src/context.h"

const int BATCH_SIZE = 8192;
const int NUM_HASHING_ROUNDS = 1;

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = (call); \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(err); \
        } \
    } while (0)

extern "C" void set_num_hashing_rounds(int rounds) {
    // Enforce a minimum of 1 hashing round
    int adjustedRounds = (rounds > 0) ? rounds : 1;
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(NUM_HASHING_ROUNDS), &adjustedRounds, sizeof(int)));
}

extern "C" void hash(uint8_t *challenge, uint8_t *nonce, uint64_t *out) {
    MemoryPool memPool(BATCH_SIZE);

    std::vector<uint8_t> seed(40);
    memcpy(seed.data(), challenge, 32);

    for (int i = 0; i < BATCH_SIZE; i++) {
        uint64_t nonce_offset = *((uint64_t*)nonce) + i;
        memcpy(seed.data() + 32, &nonce_offset, 8);
        memPool.ctxs[i] = hashx_alloc(HASHX_INTERPRETED);
        if (!memPool.ctxs[i] || !hashx_make(memPool.ctxs[i], seed.data(), 40)) {
            return;  // Handle errors properly
        }
    }

    int threadsPerBlock = 1024;
    int blocksPerGrid = (BATCH_SIZE * INDEX_SPACE + threadsPerBlock - 1) / threadsPerBlock;

    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    // Ensure at least one round is performed for valid hashing
    int rounds_to_execute = (NUM_HASHING_ROUNDS > 0) ? NUM_HASHING_ROUNDS : 1;
    do_hash_stage0i<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(memPool.ctxs, memPool.hash_space, rounds_to_execute);
    CUDA_CHECK(hipGetLastError());

    CUDA_CHECK(hipStreamSynchronize(stream));

    for (int i = 0; i < BATCH_SIZE; i++) {
        CUDA_CHECK(hipMemcpyAsync(out + i * INDEX_SPACE, memPool.hash_space[i], INDEX_SPACE * sizeof(uint64_t), hipMemcpyDeviceToHost, stream));
    }

    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipStreamDestroy(stream));
}

__global__ void do_hash_stage0i(hashx_ctx** ctxs, uint64_t** hash_space, int num_hashing_rounds) {
    uint32_t item = blockIdx.x * blockDim.x + threadIdx.x;
    if (item < BATCH_SIZE * INDEX_SPACE) {
        uint32_t batch_idx = item / INDEX_SPACE;
        uint32_t i = item % INDEX_SPACE;

        for (int round = 0; round < num_hashing_rounds; ++round) {
            hash_stage0i(ctxs[batch_idx], hash_space[batch_idx], i);
        }
    }
}

extern "C" void solve_all_stages(uint64_t *hashes, uint8_t *out, uint32_t *sols, int num_sets) {
    uint64_t *d_hashes;
    solver_heap *d_heaps;
    equix_solution *d_solutions;
    uint32_t *d_num_sols;

    CUDA_CHECK(hipMalloc(&d_hashes, num_sets * INDEX_SPACE * sizeof(uint64_t)));
    CUDA_CHECK(hipMalloc(&d_heaps, num_sets * sizeof(solver_heap)));
    CUDA_CHECK(hipMalloc(&d_solutions, num_sets * EQUIX_MAX_SOLS * sizeof(equix_solution)));
    CUDA_CHECK(hipMalloc(&d_num_sols, num_sets * sizeof(uint32_t)));

    equix_solution *h_solutions;
    uint32_t *h_num_sols;
    CUDA_CHECK(hipHostAlloc(&h_solutions, num_sets * EQUIX_MAX_SOLS * sizeof(equix_solution), hipHostMallocDefault));
    CUDA_CHECK(hipHostAlloc(&h_num_sols, num_sets * sizeof(uint32_t), hipHostMallocDefault));

    CUDA_CHECK(hipMemcpy(d_hashes, hashes, num_sets * INDEX_SPACE * sizeof(uint64_t), hipMemcpyHostToDevice));

    int threadsPerBlock = 1024;
    int blocksPerGrid = (num_sets + threadsPerBlock - 1) / threadsPerBlock;

    solve_all_stages_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_hashes, d_heaps, d_solutions, d_num_sols);
    CUDA_CHECK(hipGetLastError());

    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(h_solutions, d_solutions, num_sets * EQUIX_MAX_SOLS * sizeof(equix_solution), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_num_sols, d_num_sols, num_sets * sizeof(uint32_t), hipMemcpyDeviceToHost));

    for (int i = 0; i < num_sets; i++) {
        sols[i] = h_num_sols[i];
        if (h_num_sols[i] > 0) {
            memcpy(out + i * sizeof(equix_solution), &h_solutions[i * EQUIX_MAX_SOLS], sizeof(equix_solution));
        }
    }

    CUDA_CHECK(hipFree(d_hashes));
    CUDA_CHECK(hipFree(d_heaps));
    CUDA_CHECK(hipFree(d_solutions));
    CUDA_CHECK(hipFree(d_num_sols));

    CUDA_CHECK(hipHostFree(h_solutions));
    CUDA_CHECK(hipHostFree(h_num_sols));
}